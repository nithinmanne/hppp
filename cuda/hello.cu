#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  printf("i: %d\n", i);
  if(i<n)
    C[i] = A[i] + B[i];
}

void vecAdd(float *A, float *B, float *C, int n) {
  int s = n*sizeof(float);
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, s);
  hipMalloc((void**)&d_B, s);
  hipMalloc((void**)&d_C, s);
  hipMemcpy(d_A, A, s, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, s, hipMemcpyHostToDevice);

  vecAddKernel<<<ceil(n/2.0), 2>>>(d_A, d_B, d_C, n);

  hipMemcpy(C, d_C, s, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

int main() {
  float A[]={1,2,3,4,5};
  float B[]={2,3,4,5,6};
  float C[5];
  vecAdd(A,B,C,5);
  for(int i=0;i<5;i++) {
    printf("C[%d]=%f\n", i, C[i]);
  }
}
