#include "hip/hip_runtime.h"
#include "headers.h"
/**
 * Host main routine
 */

#define D1SIZE 1024
#define D2SIZE 128
const float D1CONVMASK[] = { 1, 1, 0, 1, 1 };
const float D2CONVMASK[] = { 1./8, 1./8, 1./8, 1./8, 0, 1./8, 1./8, 1./8, 1./8 };
#define MAX 100

int main() {
    srand(time(NULL));

    // 1D
    printf("1D\n");
    size_t d1size = D1SIZE*sizeof(float);
    float *d1h_A, *d1d_A, *d1d_C, *d1h_O, *d1d_O;
    d1h_A = (float*)malloc(d1size);
    for(int i=0;i<D1SIZE;i++) d1h_A[i] = rand()%MAX;
    d1h_O = (float*)malloc(d1size);
    hipMalloc((void**)&d1d_A, d1size);
    hipMalloc((void**)&d1d_C, 5*sizeof(float));
    hipMalloc((void**)&d1d_O, d1size);
    hipMemcpy(d1d_A, d1h_A, d1size, hipMemcpyHostToDevice);
    hipMemcpy(d1d_C, D1CONVMASK, 5*sizeof(float), hipMemcpyHostToDevice);
    int d1b = (int)ceil(D1SIZE/1024.);
    d1conv<<<d1b, 1024>>>(d1d_A, d1d_C, d1d_O, D1SIZE);
    hipMemcpy(d1h_O, d1d_O, d1size, hipMemcpyDeviceToHost);
    printf("Original Array\n");
    for(int i=0;i<D1SIZE;i++) printf("%f\n", d1h_A[i]);
    printf("Modified Array\n");
    for(int i=0;i<D1SIZE;i++) printf("%f\n", d1h_O[i]);

    //2D
    printf("2D\n");
    size_t d2size = D2SIZE*D2SIZE*sizeof(float);
    float *d2h_A, *d2d_A, *d2d_C, *d2h_O, *d2d_O;
    d2h_A = (float*)malloc(d2size);
    for(int i=0;i<D2SIZE*D2SIZE;i++) d2h_A[i] = rand()%MAX;
    d2h_O = (float*)malloc(d2size);
    hipMalloc((void**)&d2d_A, d2size);
    hipMalloc((void**)&d2d_C, 9*sizeof(float));
    hipMalloc((void**)&d2d_O, d2size);
    hipMemcpy(d2d_A, d2h_A, d2size, hipMemcpyHostToDevice);
    hipMemcpy(d2d_C, D2CONVMASK, 9*sizeof(float), hipMemcpyHostToDevice);
    int d2b = (int)ceil(D2SIZE*D2SIZE/1024.);
    d2conv<<<d2b, 1024>>>(d2d_A, d2d_C, d2d_O, D2SIZE);
    hipMemcpy(d2h_O, d2d_O, d2size, hipMemcpyDeviceToHost);
    printf("Original Array\n");
    for(int i=0;i<D2SIZE;i++,printf("\n"))
        for(int j=0;j<D2SIZE;j++)
            printf("%f\t", d2h_A[i*D2SIZE+j]);
    printf("Modified Array\n");
    for(int i=0;i<D2SIZE;i++,printf("\n"))
        for(int j=0;j<D2SIZE;j++)
            printf("%f\t", d2h_O[i*D2SIZE+j]);
}
