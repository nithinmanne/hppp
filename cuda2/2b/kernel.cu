
#include <hip/hip_runtime.h>
#include <stdio.h>
/**
 * Kernel routine
 */
__global__
void d1conv(const float* a, const float *c, float *o, const int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id<size) {
        float co = 0;
        for(int i=id-2;i<=id+2;i++)
            if(i>=0&&i<size)
                co += a[i]*c[i-id+2];
        o[id] = co;
    }
}

__global__
void d2conv(const float* a, const float *c, float *o, const int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id<size*size) {
        int idi = id/size;
        int idj = id%size;
        float co = 0;
        for(int i=idi-1;i<=idi+1;i++)
            if(i>=0&&i<size)
                for(int j=idj-1;j<=idj+1;j++)
                    if(j>=0&&j<size)
                        co += a[i*size+j]*c[(i-idi+1)*size+(j-idj+1)];
        o[idi*size+idj] = co;
    }
}
