#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include "headers.h"
/**
 * Host main routine
 */
int main(void) {

    srand(time(NULL));

    dim3 grid1(4, 2, 2);
    dim3 block1(32, 32, 1);
    int maxNumElements = 4*2*2*32*32*1;

    int remaining2 = maxNumElements/(2*1*8*8), missing2grid = 0;
    while(missing2grid==0) {
      int tmp = 1 + rand()%remaining2;
      if(remaining2%tmp!=0) continue;
      if(2*tmp*1>1024||8*8*remaining2/tmp>1024) continue;
      missing2grid = tmp;
    }
    dim3 grid2(2, missing2grid, 1);
    dim3 block2(8, 8, remaining2/missing2grid);

    int remaining3 = maxNumElements/(1*1*128*1), missing3grid = 0;
    while(missing3grid==0) {
      int tmp = 1 + rand()%remaining3;
      if(remaining3%tmp!=0) continue;
      if(tmp*1*1>1024||128*remaining3/tmp*1>1024) continue;
      missing3grid = tmp;
    }
    dim3 grid3(missing3grid, 1, 1);
    dim3 block3(128, remaining3/missing3grid, 1);

    int numElements = 1 + rand()%maxNumElements;
    printf("[Linear Transformations on arrays of %d elements]\n", numElements);
    printf("CUDA kernel 1 launch with (%d,%d,%d) blocks of (%d,%d,%d) threads\n", 4, 2, 2, 32, 32, 1);
    printf("CUDA kernel 2 launch with (%d,%d,%d) blocks of (%d,%d,%d) threads\n", 2, missing2grid, 1, 8, 8, remaining2/missing2grid);
    printf("CUDA kernel 3 launch with (%d,%d,%d) blocks of (%d,%d,%d) threads\n", missing3grid, 1, 1, 128, remaining3/missing3grid, 1);


    size_t size = numElements * sizeof(float);
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *h_aC = (float*)malloc(size);

    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        if(log(sin(h_A[i])+cos(h_B[i]))<0)
          i--;
        else
          h_aC[i] = sqrt(log(sin(h_A[i])+cos(h_B[i])));
    }

    float *d_A, *d_B, *d_C1, *d_C2, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C1, size);
    hipMalloc((void**)&d_C2, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    process_kernel1<<<grid1, block1>>>(d_A, d_B, d_C1, numElements);
    process_kernel2<<<grid2, block2>>>(d_C1, d_C2, numElements);
    process_kernel3<<<grid3, block3>>>(d_C2, d_C, numElements);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < numElements; ++i)
    {
      if (fabs(h_aC[i] - h_C[i]) > 1e-5)
      {
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          fprintf(stderr, "%f\t%f\tActual %f, Calculated %f!\n", h_A[i], h_B[i], h_aC[i], h_C[i]);
          exit(EXIT_FAILURE);
      }
    }

    printf("Test PASSED\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C1);
    hipFree(d_C2);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    struct hipDeviceProp_t test;
    hipGetDeviceProperties(&test, 0);
    printf("Device Name: %s\n", test.name);





    hipDeviceReset();
    return 0;
}
