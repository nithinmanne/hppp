
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__
int geti() {
  int i = blockIdx.z;
  i = i*gridDim.y + blockIdx.y;
  i = i*gridDim.x + blockIdx.x;
  i = i*blockDim.z + threadIdx.z;
  i = i*blockDim.y + threadIdx.y;
  i = i*blockDim.x + threadIdx.x;
  return i;
}

__global__
void process_kernel1(const float *A, const float *B, float *C, const int numElements)
{
    int i = geti();

    if (i < numElements)
    {
        C[i] = sin(A[i]) + cos(B[i]);
    }
}

__global__
void process_kernel2(const float *A, float *C, const int numElements)
{
    int i = geti();

    if (i < numElements)
    {
        C[i] = log(A[i]);
    }
}

__global__
void process_kernel3(const float *A, float *C, const int numElements)
{
    int i = geti();

    if (i < numElements)
    {
        C[i] = sqrt(A[i]);
    }
}
