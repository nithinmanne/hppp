
#include <hip/hip_runtime.h>
#include <stdio.h>
/**
 * Kernel routine
 */
__global__
void matrixSync(int* a, int size) {

    int id = threadIdx.x;

    if(id<size*size/2) {
        int i = id%size;
        int j = 2*(id/size);
        if(j+1!=size) {
            int tmp = a[i*size+j+1];
            a[i*size+j+1]=a[i*size+j];
            a[i*size+j]=tmp;
        }
    }
    __syncthreads();
    if(id<size*size) {
        int i = id%size;
        int j = id/size;
        if(i>j) {
            a[j*size+i] = a[i*size+j];
        }
    }

}
