#include "headers.h"
/**
 * Host main routine
 */

#define SIZE 32
#define NUM_MAX 100000

int main() {
    srand(time(NULL));
    int *h_A, *d_A;
    size_t size = SIZE*SIZE*sizeof(int);
    h_A = (int*)malloc(size);
    hipMalloc((void**)&d_A, size);
    printf("Original Array\n");
    for(int i=0;i<SIZE;i++) {
        for(int j=0;j<SIZE;j++) {
            h_A[i*SIZE+j] = rand()%NUM_MAX;
            printf("%d\t", h_A[i*SIZE+j]);
        }
        printf("\n");
    }
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    matrixSync<<<1, 1024>>>(d_A, SIZE);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    printf("Modified Array\n");
    for(int i=0;i<SIZE;i++) {
        for(int j=0;j<SIZE;j++)
            printf("%d\t", h_A[i*SIZE+j]);
        printf("\n");
    }
}
